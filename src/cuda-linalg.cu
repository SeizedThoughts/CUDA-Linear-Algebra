
#include <hip/hip_runtime.h>
#include <stdio.h>

int cudaReady(){
    int deviceCount;

    hipGetDeviceCount(&deviceCount);

    if(deviceCount == 0) return 0;
    
    return 1;
}

//dot product
__global__ void cudaDotProduct(double *a, double *b, int aY, int aX, int bY, int bX, double *c){
    //perform vector dot product in one thread
    /*
        aX = bY
        __                     __     __                     __     __                                                                                                  __
        | a[1][1] ... a[1][aX]  |     | b[1][1] ... b[1][bX]  |     | a[1][1] * b[1][1] + ... + a[1][aX] * b[1][bX] ... a[1][aX] * b[1][1] + ... + a[aY][aX] * b[1][bX]  |
        |     .  .        .     |     |     .  .        .     |     |                      .                       .                          .                          |
        |     .    .      .     |  *  |     .    .      .     |  =  |                      .                         .                        .                          |
        |     .      .    .     |     |     .      .    .     |     |                      .                           .                      .                          |
        |_a[aY][1]... a[aY][aX]_|     |_b[bY][1]... b[bY][bX]_|     |_a[1][1] * b[bY][1]+ ... + a[1][aX] * b[bY][bX]... a[1][aX] * b[bY][1]+ ... + a[aY][aX] * b[bY][bX]_|
    */

    int sA = aX * threadIdx.y;
    int sB = threadIdx.x;

    c[bX * threadIdx.y + threadIdx.x] = 0;

    int iA = sA;
    int iB = sB;

    while(iB < bY * bX){
        c[bX * threadIdx.y + threadIdx.x] += a[iA] * b[iB];
        iA++;
        iB += bX;
    }
}

void dotProduct(double *a, double *b, int aY, int aX, int bY, int bX, double *c){
    double *d_a, *d_b, *d_c;
    
    hipMalloc(&d_a, aY * aX * sizeof(double));
    hipMalloc(&d_b, bY * bX * sizeof(double));
    hipMalloc(&d_c, aY * bX * sizeof(double));

    hipMemcpy(d_a, a, aY * aX * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bY * bX * sizeof(double), hipMemcpyHostToDevice);
    
    dim3 threads(bX, aY);

    cudaDotProduct<<<1, threads>>>(d_a, d_b, aY, aX, bY, bX, d_c);

    hipMemcpy(c, d_c, aY * bX * sizeof(double), hipMemcpyDeviceToHost);
}

__global__ void cudaSumMatrices(double *a, double *b, int y, int x, double *c){
    int i = x * threadIdx.y + threadIdx.x;

    c[i] = a[i] + b[i];
}

void sumMatrices(double *a, double *b, int y, int x, double *c){
    double *d_a, *d_b, *d_c;
    
    hipMalloc(&d_a, y * x * sizeof(double));
    hipMalloc(&d_b, y * x * sizeof(double));
    hipMalloc(&d_c, y * x * sizeof(double));

    hipMemcpy(d_a, a, y * x * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, y * x * sizeof(double), hipMemcpyHostToDevice);
    
    dim3 threads(x, y);

    cudaSumMatrices<<<1, threads>>>(d_a, d_b, y, x, d_c);

    hipMemcpy(c, d_c, y * x * sizeof(double), hipMemcpyDeviceToHost);
}